
#include <hip/hip_runtime.h>
#ifdef SPIRIT_USE_CUDA

#include <engine/Vectormath.hpp>
#include <utility/Constants.hpp>
#include <utility/Logging.hpp>
#include <utility/Exception.hpp>

#include <Eigen/Dense>

#include <iostream>
#include <stdio.h>
#include <algorithm>

#include <curand.h>
#include <curand_kernel.h>

using namespace Utility;
using Utility::Constants::Pi;


/*
allowed:   arch <  7 and toolkit <  9   ->   no shfl_sync and not needed
allowed:   arch <  7 and toolkit >= 9   ->   shfl_sync not needed but available (non-sync is deprecated)
allowed:   arch >= 7 and toolkit >= 9   ->   shfl_sync needed and available
forbidden: arch <  7 and toolkit >= 11  ->   likely removed shfl without sync
forbidden: arch >= 7 and toolkit <  9   ->   shfl_sync needed but not available
*/
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700) && (CUDART_VERSION >= 11000)
    #error "When compiling for compute capability < 7.0, this code requires CUDA Toolkit version < 11.0"
#endif
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 700) && (CUDART_VERSION < 9000)
    #error "When compiling for compute capability >= 7.0, this code requires CUDA Toolkit version >= 9.0"
#endif


// CUDA Version
namespace Engine
{
    namespace Vectormath
    {
        /////////////////////////////////////////////////////////////////
        // BOILERPLATE CUDA Reductions

        __inline__ __device__
        scalar warpReduceSum(scalar val)
        {
            #if (CUDART_VERSION >= 9000)
            for (int offset = warpSize/2; offset > 0; offset /= 2)
                val += __shfl_down_sync(0xffffffff, val, offset);
            #else
            for (int offset = warpSize/2; offset > 0; offset /= 2)
                val += __shfl_down(val, offset);
            #endif
            return val;
        }

        __inline__ __device__
        scalar blockReduceSum(scalar val)
        {
            static __shared__ scalar shared[32]; // Shared mem for 32 partial sums
            int lane = threadIdx.x % warpSize;
            int wid = threadIdx.x / warpSize;

            val = warpReduceSum(val);     // Each warp performs partial reduction

            if (lane==0) shared[wid]=val; // Write reduced value to shared memory

            __syncthreads();              // Wait for all partial reductions

            //read from shared memory only if that warp existed
            val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

            if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

            return val;
        }

        __global__ void cu_sum(const scalar *in, scalar* out, int N)
        {
            scalar sum = int(0);
            for(int i = blockIdx.x * blockDim.x + threadIdx.x;
                i < N;
                i += blockDim.x * gridDim.x)
            {
                sum += in[i];
            }
            sum = blockReduceSum(sum);
            if (threadIdx.x == 0)
                atomicAdd(out, sum);
        }



        __inline__ __device__
        Vector3 warpReduceSum(Vector3 val)
        {
            #if (CUDART_VERSION >= 9000)
            for (int offset = warpSize/2; offset > 0; offset /= 2)
            {
                val[0] += __shfl_down_sync(0xffffffff, val[0], offset);
                val[1] += __shfl_down_sync(0xffffffff, val[1], offset);
                val[2] += __shfl_down_sync(0xffffffff, val[2], offset);
            }
            #else
            for (int offset = warpSize/2; offset > 0; offset /= 2)
            {
                val[0] += __shfl_down(val[0], offset);
                val[1] += __shfl_down(val[1], offset);
                val[2] += __shfl_down(val[2], offset);
            }
            #endif
            return val;
        }

        __inline__ __device__
        Vector3 blockReduceSum(Vector3 val)
        {
            static __shared__ Vector3 shared[32]; // Shared mem for 32 partial sums
            int lane = threadIdx.x % warpSize;
            int wid = threadIdx.x / warpSize;

            val = warpReduceSum(val);     // Each warp performs partial reduction

            if (lane==0) shared[wid]=val; // Write reduced value to shared memory

            __syncthreads();              // Wait for all partial reductions

            // Read from shared memory only if that warp existed
            val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : Vector3{0,0,0};

            if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

            return val;
        }

        __global__ void cu_sum(const Vector3 *in, Vector3* out, int N)
        {
            Vector3 sum{0,0,0};
            for(int i = blockIdx.x * blockDim.x + threadIdx.x;
                i < N;
                i += blockDim.x * gridDim.x)
            {
                sum += in[i];
            }
            sum = blockReduceSum(sum);
            if (threadIdx.x == 0)
            {
                atomicAdd(&out[0][0], sum[0]);
                atomicAdd(&out[0][1], sum[1]);
                atomicAdd(&out[0][2], sum[2]);
            }
        }


        __inline__ __device__
        scalar warpReduceMin(scalar val)
        {
            #if (CUDART_VERSION >= 9000)
            for (int offset = warpSize/2; offset > 0; offset /= 2)
                val  = min(val, __shfl_down_sync(0xffffffff, val, offset));
            #else
            for (int offset = warpSize/2; offset > 0; offset /= 2)
                val  = min(val, __shfl_down(val, offset));
            #endif
            return val;
        }
        __inline__ __device__
        scalar warpReduceMax(scalar val)
        {
            #if (CUDART_VERSION >= 9000)
            for (int offset = warpSize/2; offset > 0; offset /= 2)
                val = max(val, __shfl_down_sync(0xffffffff, val, offset));
            #else
            for (int offset = warpSize/2; offset > 0; offset /= 2)
                val = max(val, __shfl_down(val, offset));
            #endif
            return val;
        }

        __inline__ __device__
        void blockReduceMinMax(scalar val, scalar *out_min, scalar *out_max)
        {
            static __shared__ scalar shared_min[32]; // Shared mem for 32 partial minmax comparisons
            static __shared__ scalar shared_max[32]; // Shared mem for 32 partial minmax comparisons

            int lane = threadIdx.x % warpSize;
            int wid = threadIdx.x / warpSize;

            scalar _min = warpReduceMin(val);  // Each warp performs partial reduction
            scalar _max = warpReduceMax(val);  // Each warp performs partial reduction

            if (lane==0) shared_min[wid]=_min;  // Write reduced minmax to shared memory
            if (lane==0) shared_max[wid]=_max;  // Write reduced minmax to shared memory
            __syncthreads();                      // Wait for all partial reductions

            // Read from shared memory only if that warp existed
            _min  = (threadIdx.x < blockDim.x / warpSize) ? shared_min[lane] : 0;
            _max  = (threadIdx.x < blockDim.x / warpSize) ? shared_max[lane] : 0;

            if (wid==0) _min  = warpReduceMin(_min);  // Final minmax reduce within first warp
            if (wid==0) _max  = warpReduceMax(_max);  // Final minmax reduce within first warp

            out_min[0] = _min;
            out_max[0] = _max;
        }

        __global__ void cu_MinMax(const scalar *in, scalar* out_min, scalar* out_max, int N)
        {
            scalar tmp, tmp_min{0}, tmp_max{0};
            scalar _min{0}, _max{0};
            for(int i = blockIdx.x * blockDim.x + threadIdx.x;
                i < N;
                i += blockDim.x * gridDim.x)
            {
                _min = min(_min, in[i]);
                _max = max(_max, in[i]);
            }

            tmp_min = _min;
            tmp_max = _max;

            blockReduceMinMax(tmp_min, &_min, &tmp);
            blockReduceMinMax(tmp_max, &tmp, &_max);

            if (threadIdx.x==0)
            {
                out_min[blockIdx.x] = _min;
                out_max[blockIdx.x] = _max;
            }
        }

        std::pair<scalar, scalar> minmax_component(const vectorfield & vf)
        {
            int N = 3*vf.size();
            int threads = 512;
            int blocks = min((N + threads - 1) / threads, 1024);

            static scalarfield out_min(blocks, 0);
            Vectormath::fill(out_min, 0);
            static scalarfield out_max(blocks, 0);
            Vectormath::fill(out_max, 0);
            static scalarfield temp(1, 0);
            Vectormath::fill(temp, 0);

            cu_MinMax<<<blocks, threads>>>(&vf[0][0], out_min.data(), out_max.data(), N);
            cu_MinMax<<<1, 1024>>>(out_min.data(), out_min.data(), temp.data(), blocks);
            cu_MinMax<<<1, 1024>>>(out_max.data(), temp.data(), out_max.data(), blocks);
            CU_CHECK_AND_SYNC();

            return std::pair<scalar, scalar>{out_min[0], out_max[0]};
        }


        /////////////////////////////////////////////////////////////////




        // Utility function for the SIB Solver
        __global__ void cu_transform(const Vector3 * spins, const Vector3 * force, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            Vector3 e1, a2, A;
            scalar detAi;
            if(idx < N)
            {
                e1 = spins[idx];
                A = 0.5 * force[idx];

                // 1/determinant(A)
                detAi = 1.0 / (1 + pow(A.norm(), 2));

                // calculate equation without the predictor?
                a2 = e1 - e1.cross(A);

                out[idx][0] = (a2[0] * (A[0] * A[0] + 1   ) + a2[1] * (A[0] * A[1] - A[2]) + a2[2] * (A[0] * A[2] + A[1])) * detAi;
                out[idx][1] = (a2[0] * (A[1] * A[0] + A[2]) + a2[1] * (A[1] * A[1] + 1   ) + a2[2] * (A[1] * A[2] - A[0])) * detAi;
                out[idx][2] = (a2[0] * (A[2] * A[0] - A[1]) + a2[1] * (A[2] * A[1] + A[0]) + a2[2] * (A[2] * A[2] + 1   )) * detAi;
            }
        }
        void transform(const vectorfield & spins, const vectorfield & force, vectorfield & out)
        {
            int n = spins.size();
            cu_transform<<<(n+1023)/1024, 1024>>>(spins.data(), force.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }

        void get_random_vector(std::uniform_real_distribution<scalar> & distribution, std::mt19937 & prng, Vector3 & vec)
        {
            for (int dim = 0; dim < 3; ++dim)
            {
                vec[dim] = distribution(prng);
            }
        }

        __global__ void cu_get_random_vectorfield(Vector3 * xi, size_t N)
        {
            unsigned long long subsequence = 0;
            unsigned long long offset= 0;

            curandState_t state;
            for(int idx = blockIdx.x * blockDim.x + threadIdx.x;
                idx < N;
                idx +=  blockDim.x * gridDim.x)
            {
                curand_init(idx,subsequence,offset,&state);
                for (int dim=0;dim<3; ++dim)
                {
                    xi[idx][dim] = llroundf(curand_uniform(&state))*2-1;
                }
            }
        }
        void get_random_vectorfield(std::uniform_real_distribution<scalar> & distribution, std::mt19937 & prng, vectorfield & xi)
        {
            int n = xi.size();
            cu_get_random_vectorfield<<<(n+1023)/1024, 1024>>>(xi.data(), n);
            CU_CHECK_AND_SYNC();
        }

        void get_random_vector_unitsphere(std::uniform_real_distribution<scalar> & distribution, std::mt19937 & prng, Vector3 & vec)
        {
            scalar v_z = distribution(prng);
            scalar phi = distribution(prng);

            scalar r_xy = std::sqrt(1 - v_z*v_z);

            vec[0] = r_xy * std::cos(2*Pi*phi);
            vec[1] = r_xy * std::sin(2*Pi*phi);
            vec[2] = v_z;
        }
        // __global__ void cu_get_random_vectorfield_unitsphere(Vector3 * xi, size_t N)
        // {
        //     unsigned long long subsequence = 0;
        //     unsigned long long offset= 0;

        //     curandState_t state;
        //     for(int idx = blockIdx.x * blockDim.x + threadIdx.x;
        //         idx < N;
        //         idx +=  blockDim.x * gridDim.x)
        //     {
        //         curand_init(idx,subsequence,offset,&state);

        //         scalar v_z = llroundf(curand_uniform(&state))*2-1;
        //         scalar phi = llroundf(curand_uniform(&state))*2-1;

        // 	    scalar r_xy = std::sqrt(1 - v_z*v_z);

        //         xi[idx][0] = r_xy * std::cos(2*Pi*phi);
        //         xi[idx][1] = r_xy * std::sin(2*Pi*phi);
        //         xi[idx][2] = v_z;
        //     }
        // }
        // void get_random_vectorfield_unitsphere(std::mt19937 & prng, vectorfield & xi)
        // {
        //     int n = xi.size();
        //     cu_get_random_vectorfield<<<(n+1023)/1024, 1024>>>(xi.data(), n);
        //     CU_CHECK_AND_SYNC();
        // }
        // The above CUDA implementation does not work correctly.
        void get_random_vectorfield_unitsphere(std::mt19937 & prng, vectorfield & xi)
        {
            // PRNG gives RN [-1,1] -> multiply with epsilon
            auto distribution = std::uniform_real_distribution<scalar>(-1, 1);
            // TODO: parallelization of this is actually not quite so trivial
            #pragma omp parallel for
            for (unsigned int i = 0; i < xi.size(); ++i)
            {
                get_random_vector_unitsphere(distribution, prng, xi[i]);
            }
        }

       
        /////////////////////////////////////////////////////////////////


        __global__ void cu_fill(scalar *sf, scalar s, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                sf[idx] = s;
            }
        }
        void fill(scalarfield & sf, scalar s)
        {
            int n = sf.size();
            cu_fill<<<(n+1023)/1024, 1024>>>(sf.data(), s, n);
            CU_CHECK_AND_SYNC();
        }
        __global__ void cu_fill_mask(scalar *sf, scalar s, const int * mask, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                sf[idx] = mask[idx]*s;
            }
        }
        void fill(scalarfield & sf, scalar s, const intfield & mask)
        {
            int n = sf.size();
            cu_fill_mask<<<(n+1023)/1024, 1024>>>(sf.data(), s, mask.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_scale(scalar *sf, scalar s, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                sf[idx] *= s;
            }
        }
        void scale(scalarfield & sf, scalar s)
        {
            int n = sf.size();
            cu_scale<<<(n+1023)/1024, 1024>>>(sf.data(), s, n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_add(scalar *sf, scalar s, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                sf[idx] += s;
            }
        }
        void add(scalarfield & sf, scalar s)
        {
            int n = sf.size();
            cu_add<<<(n+1023)/1024, 1024>>>(sf.data(), s, n);
            cudaDeviceSynchronize();
        }

        scalar sum(const scalarfield & sf)
        {
            int N = sf.size();
            int threads = 512;
            int blocks = min((N + threads - 1) / threads, 1024);

            static scalarfield ret(1, 0);
            Vectormath::fill(ret, 0);
            cu_sum<<<blocks, threads>>>(sf.data(), ret.data(), N);
            CU_CHECK_AND_SYNC();
            return ret[0];
        }

        scalar mean(const scalarfield & sf)
        {
            int N = sf.size();
            int threads = 512;
            int blocks = min((N + threads - 1) / threads, 1024);

            static scalarfield ret(1, 0);
            Vectormath::fill(ret, 0);

            cu_sum<<<blocks, threads>>>(sf.data(), ret.data(), N);
            CU_CHECK_AND_SYNC();

            ret[0] = ret[0]/N;
            return ret[0];
        }

        __global__ void cu_divide(const scalar * numerator, const scalar * denominator, scalar * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] += numerator[idx] / denominator[idx];
            }
        }
        void divide( const scalarfield & numerator, const scalarfield & denominator, scalarfield & out )
        {
            int n = numerator.size();
            cu_divide<<<(n+1023)/1024, 1024>>>(numerator.data(), denominator.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }

        void set_range(scalarfield & sf, scalar sf_min, scalar sf_max)
        {
            #pragma omp parallel for
            for (unsigned int i = 0; i<sf.size(); ++i)
                sf[i] = std::min( std::max( sf_min, sf[i] ), sf_max );
        }

        __global__ void cu_fill(Vector3 *vf1, Vector3 v2, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                vf1[idx] = v2;
            }
        }
        void fill(vectorfield & vf, const Vector3 & v)
        {
            int n = vf.size();
            cu_fill<<<(n+1023)/1024, 1024>>>(vf.data(), v, n);
            CU_CHECK_AND_SYNC();
        }
        __global__ void cu_fill_mask(Vector3 *vf1, Vector3 v2, const int * mask, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                vf1[idx] = v2;
            }
        }
        void fill(vectorfield & vf, const Vector3 & v, const intfield & mask)
        {
            int n = vf.size();
            cu_fill_mask<<<(n+1023)/1024, 1024>>>(vf.data(), v, mask.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_normalize_vectors(Vector3 *vf, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                vf[idx].normalize();
            }
        }
        void normalize_vectors(vectorfield & vf)
        {
            int n = vf.size();
            cu_normalize_vectors<<<(n+1023)/1024, 1024>>>(vf.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_norm(const Vector3 * vf, scalar * norm, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                norm[idx] = vf[idx].norm();
            }
        }
        void norm( const vectorfield & vf, scalarfield & norm )
        {
            int n = vf.size();
            cu_norm<<<(n+1023)/1024, 1024>>>(vf.data(), norm.data(), n);
            CU_CHECK_AND_SYNC();
        }

        scalar max_abs_component(const vectorfield & vf)
        {
            // We want the Maximum of Absolute Values of all force components on all images
            // Find minimum and maximum values
            std::pair<scalar,scalar> minmax = minmax_component(vf);
            scalar absmin = std::abs(minmax.first);
            scalar absmax = std::abs(minmax.second);
            // Maximum of absolute values
            return std::max(absmin, absmax);
        }

        __global__ void cu_scale(Vector3 *vf1, scalar sc, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                vf1[idx] *= sc;
            }
        }
        void scale(vectorfield & vf, const scalar & sc)
        {
            int n = vf.size();
            cu_scale<<<(n+1023)/1024, 1024>>>(vf.data(), sc, n);
            CU_CHECK_AND_SYNC();
        }

        Vector3 sum(const vectorfield & vf)
        {
            int N = vf.size();
            int threads = 512;
            int blocks = min((N + threads - 1) / threads, 1024);

            static vectorfield ret(1, {0,0,0});
            Vectormath::fill(ret, {0,0,0});
            cu_sum<<<blocks, threads>>>(vf.data(), ret.data(), N);
            CU_CHECK_AND_SYNC();
            return ret[0];
        }

        Vector3 mean(const vectorfield & vf)
        {
            int N = vf.size();
            int threads = 512;
            int blocks = min((N + threads - 1) / threads, 1024);

            static vectorfield ret(1, {0,0,0});
            Vectormath::fill(ret, {0,0,0});

            cu_sum<<<blocks, threads>>>(vf.data(), ret.data(), N);
            CU_CHECK_AND_SYNC();

            ret[0] = ret[0]/N;
            return ret[0];
        }


        __global__ void cu_dot(const Vector3 *vf1, const Vector3 *vf2, scalar *out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = vf1[idx].dot(vf2[idx]);
            }
        }

        scalar dot(const vectorfield & vf1, const vectorfield & vf2)
        {
            int n = vf1.size();
            static scalarfield sf(n, 0);
            Vectormath::fill(sf, 0);
            scalar ret;

            // Dot product
            cu_dot<<<(n+1023)/1024, 1024>>>(vf1.data(), vf2.data(), sf.data(), n);
            CU_CHECK_AND_SYNC();

            // reduction
            ret = sum(sf);
            return ret;
        }

        void dot(const vectorfield & vf1, const vectorfield & vf2, scalarfield & s)
        {
            int n = vf1.size();

            // Dot product
            cu_dot<<<(n+1023)/1024, 1024>>>(vf1.data(), vf2.data(), s.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_scalardot(const scalar * s1, const scalar * s2, scalar * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = s1[idx] * s2[idx];
            }
        }
        // computes the product of scalars in s1 and s2
        // s1 and s2 are scalarfields
        void dot( const scalarfield & s1, const scalarfield & s2, scalarfield & out )
        {
            int n = s1.size();

            // Dot product
            cu_scalardot<<<(n+1023)/1024, 1024>>>(s1.data(), s2.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_cross(const Vector3 *vf1, const Vector3 *vf2, Vector3 *out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = vf1[idx].cross(vf2[idx]);
            }
        }
        // The wrapper for the calling of the actual kernel
        void cross(const vectorfield & vf1, const vectorfield & vf2, vectorfield & s)
        {
            int n = vf1.size();

            // Dot product
            cu_cross<<<(n+1023)/1024, 1024>>>(vf1.data(), vf2.data(), s.data(), n);
            CU_CHECK_AND_SYNC();
        }


        __global__ void cu_add_c_a(scalar c, Vector3 a, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] += c*a;
            }
        }
        // out[i] += c*a
        void add_c_a(const scalar & c, const Vector3 & a, vectorfield & out)
        {
            int n = out.size();
            cu_add_c_a<<<(n+1023)/1024, 1024>>>(c, a, out.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_add_c_a2(scalar c, const Vector3 * a, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] += c*a[idx];
            }
        }
        // out[i] += c*a[i]
        void add_c_a(const scalar & c, const vectorfield & a, vectorfield & out)
        {
            int n = out.size();
            cu_add_c_a2<<<(n+1023)/1024, 1024>>>(c, a.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_add_c_a2_mask(scalar c, const Vector3 * a, Vector3 * out, const int * mask, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] += c*mask[idx]*a[idx];
            }
        }
        // out[i] += c*a[i]
        void add_c_a(const scalar & c, const vectorfield & a, vectorfield & out, const intfield & mask)
        {
            int n = out.size();
            cu_add_c_a2_mask<<<(n+1023)/1024, 1024>>>(c, a.data(), out.data(), mask.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_add_c_a3(const scalar * c, const Vector3 * a, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] += c[idx]*a[idx];
            }
        }
        // out[i] += c[i]*a[i]
        void add_c_a( const scalarfield & c, const vectorfield & a, vectorfield & out )
        {
            int n = out.size();
            cu_add_c_a3<<<(n+1023)/1024, 1024>>>(c.data(), a.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }


        __global__ void cu_set_c_a(scalar c, Vector3 a, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = c*a;
            }
        }
        // out[i] = c*a
        void set_c_a(const scalar & c, const Vector3 & a, vectorfield & out)
        {
            int n = out.size();
            cu_set_c_a<<<(n+1023)/1024, 1024>>>(c, a, out.data(), n);
            CU_CHECK_AND_SYNC();
        }
        __global__ void cu_set_c_a_mask(scalar c, Vector3 a, Vector3 * out, const int * mask, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = mask[idx]*c*a;
            }
        }
        // out[i] = c*a
        void set_c_a(const scalar & c, const Vector3 & a, vectorfield & out, const intfield & mask)
        {
            int n = out.size();
            cu_set_c_a_mask<<<(n+1023)/1024, 1024>>>(c, a, out.data(), mask.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_set_c_a2(scalar c, const Vector3 * a, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = c*a[idx];
            }
        }
        // out[i] = c*a[i]
        void set_c_a(const scalar & c, const vectorfield & a, vectorfield & out)
        {
            int n = out.size();
            cu_set_c_a2<<<(n+1023)/1024, 1024>>>(c, a.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }
        __global__ void cu_set_c_a2_mask(scalar c, const Vector3 * a, Vector3 * out, const int * mask, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = mask[idx]*c*a[idx];
            }
        }
        // out[i] = c*a[i]
        void set_c_a(const scalar & c, const vectorfield & a, vectorfield & out, const intfield & mask)
        {
            int n = out.size();
            cu_set_c_a2_mask<<<(n+1023)/1024, 1024>>>(c, a.data(), out.data(), mask.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_set_c_a3(const scalar * c, const Vector3 * a, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = c[idx]*a[idx];
            }
        }
        // out[i] = c[i]*a[i]
        void set_c_a( const scalarfield & c, const vectorfield & a, vectorfield & out )
        {
            int n = out.size();
            cu_set_c_a3<<<(n+1023)/1024, 1024>>>(c.data(), a.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }


        __global__ void cu_add_c_dot(scalar c, Vector3 a, const Vector3 * b, scalar * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] += c*a.dot(b[idx]);
            }
        }
        // out[i] += c * a*b[i]
        void add_c_dot(const scalar & c, const Vector3 & a, const vectorfield & b, scalarfield & out)
        {
            int n = out.size();
            cu_add_c_dot<<<(n+1023)/1024, 1024>>>(c, a, b.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_add_c_dot(scalar c, const Vector3 * a, const Vector3 * b, scalar * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] += c*a[idx].dot(b[idx]);
            }
        }
        // out[i] += c * a[i]*b[i]
        void add_c_dot(const scalar & c, const vectorfield & a, const vectorfield & b, scalarfield & out)
        {
            int n = out.size();
            cu_add_c_dot<<<(n+1023)/1024, 1024>>>(c, a.data(), b.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }


        __global__ void cu_set_c_dot(scalar c, Vector3 a, const Vector3 * b, scalar * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = c*a.dot(b[idx]);
            }
        }
        // out[i] = c * a*b[i]
        void set_c_dot(const scalar & c, const Vector3 & a, const vectorfield & b, scalarfield & out)
        {
            int n = out.size();
            cu_set_c_dot<<<(n+1023)/1024, 1024>>>(c, a, b.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }

        __global__ void cu_set_c_dot(scalar c, const Vector3 * a, const Vector3 * b, scalar * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = c*a[idx].dot(b[idx]);
            }
        }
        // out[i] = c * a[i]*b[i]
        void set_c_dot(const scalar & c, const vectorfield & a, const vectorfield & b, scalarfield & out)
        {
            int n = out.size();
            cu_set_c_dot<<<(n+1023)/1024, 1024>>>(c, a.data(), b.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }


        // out[i] += c * a x b[i]
        __global__ void cu_add_c_cross(scalar c, const Vector3 a, const Vector3 * b, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] += c*a.cross(b[idx]);
            }
        }
        void add_c_cross(const scalar & c, const Vector3 & a, const vectorfield & b, vectorfield & out)
        {
            int n = out.size();
            cu_add_c_cross<<<(n+1023)/1024, 1024>>>(c, a, b.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }

        // out[i] += c * a[i] x b[i]
        __global__ void cu_add_c_cross(scalar c, const Vector3 * a, const Vector3 * b, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] += c*a[idx].cross(b[idx]);
            }
        }
        void add_c_cross(const scalar & c, const vectorfield & a, const vectorfield & b, vectorfield & out)
        {
            int n = out.size();
            cu_add_c_cross<<<(n+1023)/1024, 1024>>>(c, a.data(), b.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }

        // out[i] += c * a[i] x b[i]
        __global__ void cu_add_c_cross(const scalar * c, const Vector3 * a, const Vector3 * b, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] += c[idx]*a[idx].cross(b[idx]);
            }
        }
        void add_c_cross(const scalarfield & c, const vectorfield & a, const vectorfield & b, vectorfield & out)
        {
            int n = out.size();
            cu_add_c_cross<<<(n+1023)/1024, 1024>>>(c.data(), a.data(), b.data(), out.data(), n);
            cudaDeviceSynchronize();
        }


        // out[i] = c * a x b[i]
        __global__ void cu_set_c_cross(scalar c, const Vector3 a, const Vector3 * b, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = c*a.cross(b[idx]);
            }
        }
        void set_c_cross(const scalar & c, const Vector3 & a, const vectorfield & b, vectorfield & out)
        {
            int n = out.size();
            cu_set_c_cross<<<(n+1023)/1024, 1024>>>(c, a, b.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }

        // out[i] = c * a[i] x b[i]
        __global__ void cu_set_c_cross(scalar c, const Vector3 * a, const Vector3 * b, Vector3 * out, size_t N)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if(idx < N)
            {
                out[idx] = c*a[idx].cross(b[idx]);
            }
        }
        void set_c_cross(const scalar & c, const vectorfield & a, const vectorfield & b, vectorfield & out)
        {
            int n = out.size();
            cu_set_c_cross<<<(n+1023)/1024, 1024>>>(c, a.data(), b.data(), out.data(), n);
            CU_CHECK_AND_SYNC();
        }
    }
}

#endif